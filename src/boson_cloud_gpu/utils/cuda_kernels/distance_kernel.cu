
#include <hip/hip_runtime.h>
/*
Copyright(C) 2023
Riccardo Felicetti(felicettiriccardo1 @gmail.com)

This program is free software : you can redistribute it and / or modify
it under the terms of the GNU AFFERO GENERAL PUBLIC LICENSE Version 3,        \
    19 November 2007

Copyright(C) 2007 Free Software Foundation, Inc.< https: // fsf.org/>
Everyone is permitted to copy and distribute verbatim copies
of this license document, but changing it is not allowed.

You should have received a copy of theGNU AFFERO GENERAL PUBLIC LICENSE
along with this program.If not, see < http: // www.gnu.org/licenses/>.
*/

extern "C" __global__ void distance(const float *position,
                                    const int n_positions, float *distance) {
  int x_abs = threadIdx.x + blockDim.x * blockIdx.x;

  if (x_abs < n_positions) {
    distance[x_abs] = sqrt(position[x_abs * 3] * position[x_abs * 3] +
                           position[x_abs * 3 + 1] * position[x_abs * 3 + 1] +
                           position[x_abs * 3 + 2] * position[x_abs * 3 + 2]);
  }
}